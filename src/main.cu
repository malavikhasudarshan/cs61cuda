#include "hip/hip_runtime.h"
#include "utils.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>

//CPU matmul declaration
void cpu_matmul(const float *A, const float *B, float *C, int m, int n, int k);

int main() {
    srand(42);  //seed for reproducibility
    int m = 512, n = 512, k = 512;
    float *A, *B, *C_cpu, *C_gpu;

    malloc_matrix(&A, m, k);
    malloc_matrix(&B, k, n);
    malloc_matrix(&C_cpu, m, n);
    malloc_matrix(&C_gpu, m, n);
    init_matrix(A, m, k);
    init_matrix(B, k, n);

    //CPU computation baseline
    std::cout << "Running CPU baseline..." << std::endl;
    clock_t startTime = clock();
    cpu_matmul(A, B, C_cpu, m, n, k);
    clock_t endTime = clock();
    std::cout << "CPU time: " << (float)(endTime - startTime) / CLOCKS_PER_SEC << "s\n";

    //GPU allocation
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(float) * m * k);
    hipMalloc(&d_B, sizeof(float) * k * n);
    hipMalloc(&d_C, sizeof(float) * m * n);

    hipMemcpy(d_A, A, sizeof(float)*m*k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float)*k*n, hipMemcpyHostToDevice);

    //naive GPU computation
    std::cout << "Running CUDA kernel..." << std::endl;
    startTime = clock();
    launch_naive_kernel(d_A, d_B, d_C, m, n, k);
    hipMemcpy(C_gpu, d_C, sizeof(float)*m*n, hipMemcpyDeviceToHost);
    endTime = clock();
    std::cout << "GPU time (naive): " << (float)(endTime - startTime) / CLOCKS_PER_SEC << "s\n";

    //for validating correctness
    bool match = validate(C_gpu, C_cpu, m, n, 1e-3f);
    std::cout << (match ? "✅ Validation passed!\n" : "❌ Validation failed.\n");

    //remember to free memory!
    free_matrix(A);
    free_matrix(B);
    free_matrix(C_cpu);
    free_matrix(C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();
    return 0;
}